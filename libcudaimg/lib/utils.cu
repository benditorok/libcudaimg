#pragma once

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>
#include "utils.cuh"

namespace utils
{
	inline void gpuAssert(hipError_t code, const char* file, int line, bool abort)
	{
		if (code != hipSuccess)
		{
			fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);

			if (abort)
			{
				hipDeviceReset();
				exit(code);
			}
		}
	}
}
