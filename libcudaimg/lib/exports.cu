#pragma once

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include "utils.cuh"
#include "exports.cuh"
#include "kernels.cuh"

using namespace utils;

namespace exports
{
	void invertImage(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height) {
		unsigned char* d_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Launch the kernel
		kernels::invertImage<<<gridSize, blockSize>>>(d_image, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
	}

	void gammaTransformImage(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height, float gamma) {
		unsigned char* d_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Launch the kernel
		kernels::gammaTransformImage<<<gridSize, blockSize>>>(d_image, width, height, gamma);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
	}

	void logarithmicTransformImage(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height, float base)
	{
		unsigned char* d_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Launch the kernel
		kernels::logarithmicTransformImage << <gridSize, blockSize >> > (d_image, width, height, base);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
	}

	void grayscaleImage(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height)
	{
		unsigned char* d_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		const uint32_t THREADS_PER_BLOCK = 256;
		uint32_t num_pixels = width * height;
		uint32_t blocks = (num_pixels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

		// Launch the kernel
        kernels::grayscaleImage<<<blocks, THREADS_PER_BLOCK >>>(d_image, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
	}

	void computeHistogram(unsigned char* image, uint32_t image_len, uint32_t* histogram, uint32_t width, uint32_t height)
	{
		unsigned char* d_image;
		uint32_t* d_histogram;
		size_t imageSize = image_len * sizeof(unsigned char);
		size_t histogramSize = 256 * sizeof(uint32_t);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));
		gpuErrchk(hipMalloc((void**)&d_histogram, histogramSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		const uint32_t THREADS_PER_BLOCK = 256;
		uint32_t num_pixels = width * height;
		uint32_t blocks = (num_pixels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

		// Launch the kernel
		kernels::computeHistogram<<<blocks, THREADS_PER_BLOCK>>>(d_image, d_histogram, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the histogram back to the host
		gpuErrchk(hipMemcpy(histogram, d_histogram, histogramSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
		gpuErrchk(hipFree(d_histogram));
	}
}
