#pragma once

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include "utils.cuh"
#include "exports.cuh"
#include "kernels.cuh"
#include <vector>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

using namespace utils;

namespace exports
{
	void invertImage(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height) {
		unsigned char* d_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Launch the kernel
		kernels::invertImage << <gridSize, blockSize >> > (d_image, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
	}

	void gammaTransformImage(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height, float gamma) {
		unsigned char* d_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Launch the kernel
		kernels::gammaTransformImage << <gridSize, blockSize >> > (d_image, width, height, gamma);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
	}

	void logarithmicTransformImage(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height, float base)
	{
		unsigned char* d_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Launch the kernel
		kernels::logarithmicTransformImage << <gridSize, blockSize >> > (d_image, width, height, base);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
	}

	void grayscaleImage(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height)
	{
		unsigned char* d_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		const uint32_t THREADS_PER_BLOCK = 256;
		uint32_t num_pixels = width * height;
		uint32_t blocks = (num_pixels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

		// Launch the kernel
		kernels::grayscaleImage<<<blocks, THREADS_PER_BLOCK >>>(d_image, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
	}

	void computeHistogram(unsigned char* image, uint32_t image_len, uint32_t* histogram, uint32_t width, uint32_t height)
	{
		unsigned char* d_image;
		uint32_t* d_histogram;
		size_t imageSize = image_len * sizeof(unsigned char);
		size_t histogramSize = 256 * sizeof(uint32_t);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));
		gpuErrchk(hipMalloc((void**)&d_histogram, histogramSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		const uint32_t THREADS_PER_BLOCK = 256;
		uint32_t num_pixels = width * height;
		uint32_t blocks = (num_pixels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

		// Launch the kernel
		kernels::computeHistogram << <blocks, THREADS_PER_BLOCK >> > (d_image, d_histogram, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the histogram back to the host
		gpuErrchk(hipMemcpy(histogram, d_histogram, histogramSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
		gpuErrchk(hipFree(d_histogram));
	}

	void balanceHistogram(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height)
	{
		unsigned char* d_image;
		unsigned char* d_output_image;
		uint32_t* d_histogram;
		float* d_cdf;

		size_t imageSize = image_len * sizeof(unsigned char);
		size_t histogramSize = 256 * sizeof(uint32_t);
		size_t cdfSize = 256 * sizeof(float);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));
		gpuErrchk(hipMalloc((void**)&d_output_image, imageSize));
		gpuErrchk(hipMalloc((void**)&d_histogram, histogramSize));
		gpuErrchk(hipMalloc((void**)&d_cdf, cdfSize));

		// Initialize histogram to zero
		gpuErrchk(hipMemset(d_histogram, 0, histogramSize));

		// Copy the input image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Step 1: Compute the histogram
		kernels::computeHistogram << <gridSize, blockSize >> > (d_image, d_histogram, width, height);
		gpuErrchk(hipGetLastError());
		gpuErrchk(hipDeviceSynchronize());

		// Step 2: Compute the CDF
		uint32_t num_pixels = width * height;
		kernels::histogram_balancing::computeCDF << <1, 256 >> > (d_histogram, d_cdf, num_pixels);
		gpuErrchk(hipGetLastError());
		gpuErrchk(hipDeviceSynchronize());

		// Step 3: Apply histogram equalization
		kernels::histogram_balancing::applyEqualization << <gridSize, blockSize >> > (d_image, d_output_image, d_cdf, width, height);
		gpuErrchk(hipGetLastError());
		gpuErrchk(hipDeviceSynchronize());

		// Step 4: Copy the result back to the original image (in-place modification)
		gpuErrchk(hipMemcpy(image, d_output_image, imageSize, hipMemcpyDeviceToHost));

		// Free allocated memory on the GPU
		gpuErrchk(hipFree(d_image));
		gpuErrchk(hipFree(d_output_image));
		gpuErrchk(hipFree(d_histogram));
		gpuErrchk(hipFree(d_cdf));
	}

	void boxFilter(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height, uint32_t filter_size)
	{
		unsigned char* d_image;
		unsigned char* d_output_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));
		gpuErrchk(hipMalloc((void**)&d_output_image, imageSize));

		// Copy the input image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Launch the kernel
		kernels::boxFilter << <gridSize, blockSize >> > (d_image, d_output_image, width, height, filter_size);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_output_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
		gpuErrchk(hipFree(d_output_image));
	}

	void gaussianBlur(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height, float sigma)
	{
		unsigned char* d_image;
		unsigned char* d_output_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));
		gpuErrchk(hipMalloc((void**)&d_output_image, imageSize));

		// Copy the input image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Launch the kernel
		kernels::gaussianBlur << <gridSize, blockSize >> > (d_image, d_output_image, width, height, sigma);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_output_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
		gpuErrchk(hipFree(d_output_image));
	}

	void sobelEdgeDetection(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height)
	{
		unsigned char* d_image;
		unsigned char* d_output_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));
		gpuErrchk(hipMalloc((void**)&d_output_image, imageSize));

		// Copy the input image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Preprocessing steps
		kernels::grayscaleImage << <(width * height + 256 - 1) / 256, 256 >> > (d_image, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		kernels::gaussianBlur << <gridSize, blockSize >> > (d_image, d_image, width, height, 1.0f);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Launch the kernel
		kernels::sobelEdgeDetection << <gridSize, blockSize >> > (d_image, d_output_image, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_output_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
		gpuErrchk(hipFree(d_output_image));
	}

	void laplaceEdgeDetection(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height)
	{
		unsigned char* d_image;
		unsigned char* d_output_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));
		gpuErrchk(hipMalloc((void**)&d_output_image, imageSize));

		// Copy the input image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Preprocessing steps
		kernels::grayscaleImage << <(width * height + 256 - 1) / 256, 256 >> > (d_image, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		kernels::gaussianBlur << <gridSize, blockSize >> > (d_image, d_image, width, height, 1.0f);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Launch the kernel
		kernels::laplaceEdgeDetection << <gridSize, blockSize >> > (d_image, d_output_image, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_output_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
		gpuErrchk(hipFree(d_output_image));
	}

	void harrisCornerDetection(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height)
	{
		unsigned char* d_image;
		unsigned char* d_output_image;
		float* d_grad_x;
		float* d_grad_y;
		float* d_response;
		float k = 0.04f;
		float threshold = 1e6f;

		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));
		gpuErrchk(hipMalloc((void**)&d_output_image, imageSize));
		gpuErrchk(hipMalloc(&d_grad_x, width * height * sizeof(float)));
		gpuErrchk(hipMalloc(&d_grad_y, width * height * sizeof(float)));
		gpuErrchk(hipMalloc(&d_response, width * height * sizeof(float)));

		// Copy the input image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize * sizeof(unsigned char), hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Preprocessing steps
		kernels::grayscaleImage << <(width * height + 256 - 1) / 256, 256 >> > (d_image, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		kernels::gaussianBlur << <gridSize, blockSize >> > (d_image, d_image, width, height, 1.0f);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Launch the kernels
		kernels::harris::computeGradients << <gridSize, blockSize >> > (d_image, d_grad_x, d_grad_y, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		kernels::harris::computeHarrisResponse << <gridSize, blockSize >> > (d_grad_x, d_grad_y, d_response, width, height, k);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		kernels::harris::nonMaxSuppression << <gridSize, blockSize >> > (d_response, d_output_image, width, height, threshold);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_output_image, imageSize, hipMemcpyDeviceToHost));

		hipFree(d_image);
		hipFree(d_output_image);
		hipFree(d_grad_x);
		hipFree(d_grad_y);
		hipFree(d_response);
	}
}
