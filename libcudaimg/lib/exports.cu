#pragma once

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include "utils.cuh"
#include "exports.cuh"
#include "kernels.cuh"

using namespace utils;

namespace exports
{
	void invertImage(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height) {
		unsigned char* d_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Launch the kernel
		kernels::invertImage<<<gridSize, blockSize>>>(d_image, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
	}

	void gammaTransformImage(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height, float gamma) {
		unsigned char* d_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Launch the kernel
		kernels::gammaTransformImage<<<gridSize, blockSize>>>(d_image, width, height, gamma);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
	}

	void logarithmicTransformImage(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height, float base)
	{
		unsigned char* d_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Launch the kernel
		kernels::logarithmicTransformImage << <gridSize, blockSize >> > (d_image, width, height, base);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
	}

	void grayscaleImage(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height)
	{
		unsigned char* d_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		const uint32_t THREADS_PER_BLOCK = 256;
		uint32_t num_pixels = width * height;
		uint32_t blocks = (num_pixels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

		// Launch the kernel
        kernels::grayscaleImage<<<blocks, THREADS_PER_BLOCK >>>(d_image, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
	}

	void computeHistogram(unsigned char* image, uint32_t image_len, uint32_t* histogram, uint32_t width, uint32_t height)
	{
		unsigned char* d_image;
		uint32_t* d_histogram;
		size_t imageSize = image_len * sizeof(unsigned char);
		size_t histogramSize = 256 * sizeof(uint32_t);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));
		gpuErrchk(hipMalloc((void**)&d_histogram, histogramSize));

		// Copy the image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		const uint32_t THREADS_PER_BLOCK = 256;
		uint32_t num_pixels = width * height;
		uint32_t blocks = (num_pixels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

		// Launch the kernel
		kernels::computeHistogram<<<blocks, THREADS_PER_BLOCK>>>(d_image, d_histogram, width, height);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the histogram back to the host
		gpuErrchk(hipMemcpy(histogram, d_histogram, histogramSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
		gpuErrchk(hipFree(d_histogram));
	}

	void balanceHistogram(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height)
	{
		unsigned char* d_image;
		unsigned char* d_output_image;
		uint32_t* d_histogram;
		float* d_cdf;

		size_t imageSize = image_len * sizeof(unsigned char);
		size_t histogramSize = 256 * sizeof(uint32_t);
		size_t cdfSize = 256 * sizeof(float);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));
		gpuErrchk(hipMalloc((void**)&d_output_image, imageSize));
		gpuErrchk(hipMalloc((void**)&d_histogram, histogramSize));
		gpuErrchk(hipMalloc((void**)&d_cdf, cdfSize));

		// Initialize histogram to zero
		gpuErrchk(hipMemset(d_histogram, 0, histogramSize));

		// Copy the input image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Step 1: Compute the histogram
		kernels::computeHistogram << <gridSize, blockSize >> > (d_image, d_histogram, width, height);
		gpuErrchk(hipGetLastError());
		gpuErrchk(hipDeviceSynchronize());

		// Step 2: Compute the CDF
		uint32_t num_pixels = width * height;
		kernels::histogram_balancing::computeCDF << <1, 256 >> > (d_histogram, d_cdf, num_pixels);
		gpuErrchk(hipGetLastError());
		gpuErrchk(hipDeviceSynchronize());

		// Step 3: Apply histogram equalization
		kernels::histogram_balancing::applyEqualization << <gridSize, blockSize >> > (d_image, d_output_image, d_cdf, width, height);
		gpuErrchk(hipGetLastError());
		gpuErrchk(hipDeviceSynchronize());

		// Step 4: Copy the result back to the original image (in-place modification)
		gpuErrchk(hipMemcpy(image, d_output_image, imageSize, hipMemcpyDeviceToHost));

		// Free allocated memory on the GPU
		gpuErrchk(hipFree(d_image));
		gpuErrchk(hipFree(d_output_image));
		gpuErrchk(hipFree(d_histogram));
		gpuErrchk(hipFree(d_cdf));
	}

	void boxFilter(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height, uint32_t filterSize)
	{
		unsigned char* d_image;
		unsigned char* d_output_image;
		size_t imageSize = image_len * sizeof(unsigned char);

		// Allocate memory on the GPU
		gpuErrchk(hipMalloc((void**)&d_image, imageSize));
		gpuErrchk(hipMalloc((void**)&d_output_image, imageSize));

		// Copy the input image to device memory
		gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

		// Define block and grid sizes
		dim3 blockSize(16, 16);
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

		// Launch the kernel
		kernels::boxFilter << <gridSize, blockSize >> > (d_image, d_output_image, width, height, filterSize);
		gpuErrchk(hipGetLastError()); // Check for kernel launch errors
		gpuErrchk(hipDeviceSynchronize());

		// Copy the processed image back to the host
		gpuErrchk(hipMemcpy(image, d_output_image, imageSize, hipMemcpyDeviceToHost));

		// Free the device memory
		gpuErrchk(hipFree(d_image));
		gpuErrchk(hipFree(d_output_image));
	}
}
