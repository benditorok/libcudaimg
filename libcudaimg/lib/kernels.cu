#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cmath>

#include "utils.cuh"

namespace kernels
{
	// Invert all the pixels in the image
	__global__ void invertImage(unsigned char* image, uint32_t width, uint32_t height)
	{
		uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
		uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x < width && y < height)
		{
			uint32_t index = y * width + x;
			image[index] = 255 - image[index]; // Invert pixel value
		}
	}

	// Apply gamma transformation to the image
	__global__ void gammaTransformImage(unsigned char* image, uint32_t width, uint32_t height, float gamma)
	{
		uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
		uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x < width && y < height)
		{
			uint32_t index = y * width + x;
			image[index] = pow(image[index] / 255.0f, gamma) * 255; // Apply gamma transformation
		}
	}

	// Apply logarithmic transformation to the image
	__global__ void logarithmicTransformImage(unsigned char* image, uint32_t width, uint32_t height, float base)
	{
		uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
		uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x < width && y < height)
		{
			uint32_t index = y * width + x;
			image[index] = logf(1 + image[index]) / logf(1 + base) * 255; // Apply logarithmic transformation
		}
	}

	// Convert the image to grayscale
	__global__ void grayscaleImage(unsigned char* image, uint32_t width, uint32_t height)
	{
		uint32_t pixel_id = blockIdx.x * blockDim.x + threadIdx.x;
		uint32_t total_pixels = width * height;
		size_t rgb_index = pixel_id * 3;

		if (rgb_index < total_pixels)
		{
			unsigned char& r = image[rgb_index];
			unsigned char& g = image[rgb_index + 1];
			unsigned char& b = image[rgb_index + 2];

			// Calculate the grayscale value
			unsigned char gray = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);

			image[rgb_index] = gray;
			image[rgb_index + 1] = gray;
			image[rgb_index + 2] = gray;
		}
	}

	// Compute the histogram of the image
	__global__ void computeHistogram(unsigned char* image, uint32_t* histogram, uint32_t width, uint32_t height)
	{
		uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
		uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x < width && y < height) {
			uint32_t index = y * width + x;
			unsigned char& pixelValue = image[index];

			atomicAdd(&histogram[pixelValue], 1);
		}
	}

	namespace histogram_balancing
	{
		__global__ void computeCDF(const uint32_t* hist, float* cdf, uint32_t num_pixels)
		{
			__shared__ float shared_cdf[256];

			int idx = threadIdx.x;

			if (idx < 256) 
			{
				shared_cdf[idx] = 0;

				if (idx == 0) 
				{
					// Initialize the first value of CDF
					shared_cdf[0] = (float)hist[0] / num_pixels;

					// Calculate the cumulative sum
					for (int i = 1; i < 256; ++i) 
					{
						shared_cdf[i] = shared_cdf[i - 1] + (float)hist[i] / num_pixels;
					}
				}
			}

			__syncthreads();

			// Copy to global memory
			if (idx < 256) 
			{
				cdf[idx] = shared_cdf[idx];
			}
		}

		__global__ void applyEqualization(const unsigned char* input_img, unsigned char* output_img, const float* cdf, uint32_t width, uint32_t height) {
			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;

			if (x < width && y < height) {
				int idx = y * width + x; // Linear index from 2D coordinates
				output_img[idx] = (unsigned char)(255 * cdf[input_img[idx]]);
			}
		}
	}
}
