#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cmath>

#include "utils.cuh"

namespace kernels
{
	// Invert all the pixels in the image
	__global__ void invertImage(unsigned char* image, uint32_t width, uint32_t height)
	{
		uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
		uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x < width && y < height)
		{
			uint32_t index = y * width + x;
			image[index] = 255 - image[index]; // Invert pixel value
		}
	}

	// Apply gamma transformation to the image
	__global__ void gammaTransformImage(unsigned char* image, uint32_t width, uint32_t height, float gamma)
	{
		uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
		uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x < width && y < height)
		{
			uint32_t index = y * width + x;
			image[index] = pow(image[index] / 255.0f, gamma) * 255; // Apply gamma transformation
		}
	}

	// Apply logarithmic transformation to the image
	__global__ void logarithmicTransformImage(unsigned char* image, uint32_t width, uint32_t height, float base)
	{
		uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
		uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x < width && y < height)
		{
			uint32_t index = y * width + x;
			image[index] = logf(1 + image[index]) / logf(1 + base) * 255; // Apply logarithmic transformation
		}
	}

	// Convert the image to grayscale
	__global__ void grayscaleImage(unsigned char* image, uint32_t width, uint32_t height)
	{
		uint32_t pixel_id = blockIdx.x * blockDim.x + threadIdx.x;
		uint32_t total_pixels = width * height;
		size_t rgb_index = pixel_id * 3;

		if (rgb_index < total_pixels)
		{
			unsigned char& r = image[rgb_index];
			unsigned char& g = image[rgb_index + 1];
			unsigned char& b = image[rgb_index + 2];

			// Calculate the grayscale value
			unsigned char gray = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);

			image[rgb_index] = gray;
			image[rgb_index + 1] = gray;
			image[rgb_index + 2] = gray;
		}
	}

	__global__ void computeHistogram(unsigned char* image, uint32_t* histogram, uint32_t width, uint32_t height)
	{
		uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
		uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x < width && y < height) {
			uint32_t index = y * width + x;
			unsigned char& pixelValue = image[index];
			
			atomicAdd(&histogram[pixelValue], 1);
		}
	}
}
