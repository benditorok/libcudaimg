﻿#include "hip/hip_runtime.h"

#include <stdio.h>

// Error checking macro
#define CUDA_CHECK(call)                                                    \
    do {                                                                    \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            fprintf(stderr, "CUDA Error: %s (error code: %d)\n",            \
                    hipGetErrorString(err), err);                          \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    } while (0)

// Kernel that inverts a grayscale image
__global__ void invertImage(unsigned char* image, uint32_t width, uint32_t height) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        uint32_t index = y * width + x;
        image[index] = 255 - image[index]; // Invert pixel value
    }
}

// Exposed function that will be called from the host
extern "C" __declspec(dllexport)
void invertImage(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height) {
    unsigned char* d_image;
    size_t imageSize = image_len * sizeof(unsigned char);

    // Allocate memory on the GPU
    CUDA_CHECK(hipMalloc((void**)&d_image, imageSize));

    // Copy the image to device memory
    CUDA_CHECK(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

    // Launch the kernel
    invertImage<<<gridSize, blockSize>>>(d_image, width, height);
    CUDA_CHECK(hipGetLastError()); // Check for kernel launch errors

    // Copy the processed image back to the host
    CUDA_CHECK(hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost));

    // Free the device memory
    CUDA_CHECK(hipFree(d_image));
}
