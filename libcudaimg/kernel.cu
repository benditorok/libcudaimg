﻿#include "hip/hip_runtime.h"

#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);

        if (abort)
        {
            hipDeviceReset();
            exit(code);
        }
    }
}

// Invert all the pixels in the image
__global__ void invertImage(unsigned char* image, uint32_t width, uint32_t height) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        uint32_t index = y * width + x;
        image[index] = 255 - image[index]; // Invert pixel value
    }
}

/// <summary>
/// Exposed function which calls the invertImage kernel.
/// </summary>
/// <param name="image">Pointer to the first byte of the image.</param>
/// <param name="image_len">The number of bytes in the image.</param>
/// <param name="width">The width of the image, should be multiplied by 3 if it's in an RGB format.</param>
/// <param name="height">The height of the image.</param>
extern "C" __declspec(dllexport)
void invertImage(unsigned char* image, uint32_t image_len, uint32_t width, uint32_t height) {
    unsigned char* d_image;
    size_t imageSize = image_len * sizeof(unsigned char);

    // Allocate memory on the GPU
    gpuErrchk(hipMalloc((void**)&d_image, imageSize));

    // Copy the image to device memory
    gpuErrchk(hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice));

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);

    // Launch the kernel
    invertImage<<<gridSize, blockSize>>>(d_image, width, height);
    gpuErrchk(hipGetLastError()); // Check for kernel launch errors

    // Copy the processed image back to the host
    gpuErrchk(hipMemcpy(image, d_image, imageSize, hipMemcpyDeviceToHost));

    // Free the device memory
    gpuErrchk(hipFree(d_image));
}
